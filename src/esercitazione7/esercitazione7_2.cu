#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

typedef float *floatptr;

#define fsize(x) ((x)*sizeof(float))

void fill_matrix(floatptr vec, int m, int n) {
    for (int i = 0; i < m * n; ++i) {
        vec[i] = (float) i;
    }
}

void fill_vector(floatptr vec, int n) {
    for (int i = 0; i < n; ++i) {
        vec[i] = (float) i;
    }
}

int main(int argc, char *argv[]) {
    if (argc != 3) { exit(1); }
    int M = atoi(argv[1]);
    int N = atoi(argv[2]);
    int mat_size = M * N;
    int vec_size = N;
    hipblasHandle_t handle;
    floatptr mat_dev, vec_dev, out_dev;
    float result = 0;     // Risultato finale

    /*
    [3, 10, 20] * [5, 10, 15] = 415
    */

    floatptr mat_host = (floatptr) malloc(fsize(mat_size));      // Alloco h_a e lo inizializzo
    floatptr vec_host = (floatptr) malloc(fsize(vec_size));  // Alloco h_b e lo inizializzo
    floatptr out_host = (floatptr) malloc(fsize(vec_size));  // Alloco h_b e lo inizializzo
    fill_matrix(mat_host, M, N);
    fill_vector(vec_host, N);

    hipblasCreate(&handle);               // Creo l'handle per cublas
    hipMalloc((void **) &mat_dev, fsize(mat_size));       // Alloco d_a
    hipMalloc((void **) &vec_dev, fsize(vec_size));       // Alloco d_b
    hipMalloc((void **) &out_dev, fsize(vec_size));       // Alloco d_b
    hipblasSetMatrix(M, N, sizeof(float), mat_host, M, mat_dev, M);
    hipblasSetVector(N, sizeof(float), vec_host, 1, vec_dev, 1);
    float scalar = 1;
    float beta = 0;
    // Creazione eventi
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &scalar, mat_dev, M, vec_dev, 1, &beta, out_dev, 1);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipblasGetVector(N, sizeof(float), out_dev, 1, out_host, 1);

    if (N <= 10) {
        for (int i = 0; i < N; ++i) {
            printf("%.1f, ", out_host[i]);
        }
        printf("\n");
    }

    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Risultato del prodotto %f in %.4f ms\n", result, elapsed);

    hipFree(vec_dev);
    hipFree(out_dev);
    hipFree(mat_host);

    hipblasDestroy(handle);  // Distruggo l'handle

    free(vec_host);
    free(out_host);
    free(mat_host);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return EXIT_SUCCESS;
}