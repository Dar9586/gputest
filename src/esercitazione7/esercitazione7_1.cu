#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

typedef float *floatptr;

void fill_vector(floatptr vec, int len) {
    for (int i = 0; i < len; ++i) {
        vec[i] = (float) i;
    }
}

int main (int argc,char*argv[]){
    if(argc!=2){exit(1);}
    int M = atoi(argv[1]);
    hipblasHandle_t handle;
    floatptr h_a;         // Host array a
    floatptr d_a;         // Device array a
    floatptr h_b;         // Host array b
    floatptr d_b;         // Device array b
    float result = 0;     // Risultato finale

    /*
    [3, 10, 20] * [5, 10, 15] = 415
    */

    h_a = (floatptr)malloc (M * sizeof (*h_a));      // Alloco h_a e lo inizializzo
    h_b = (floatptr)malloc (M * sizeof (*h_b));  // Alloco h_b e lo inizializzo
    fill_vector(h_b,M);
    fill_vector(h_a,M);

    hipblasCreate(&handle);               // Creo l'handle per cublas
    hipMalloc ((void**)&d_a, M*sizeof(*h_a));       // Alloco d_a
    hipMalloc ((void**)&d_b, M*sizeof(*h_b));       // Alloco d_b
    hipblasSetVector(M,sizeof(float),h_a,1,d_a,1);    // Setto h_a su d_a
    hipblasSetVector(M,sizeof(float),h_b,1,d_b,1);    // Setto h_b su d_b

    // Creazione eventi
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipblasSdot(handle,M,d_a,1,d_b,1,&result);        // Calcolo il prodotto
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("Risultato del prodotto %f in %.4f ms\n",result,elapsed);

    hipFree (d_a);     // Dealloco d_a
    hipFree (d_b);     // Dealloco d_b

    hipblasDestroy(handle);  // Distruggo l'handle

    free(h_a);      // Dealloco h_a
    free(h_b);      // Dealloco h_b

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return EXIT_SUCCESS;
}