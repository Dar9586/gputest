#include <hip/hip_runtime.h>

#include <cstdio>
// 8 registri
__global__ void prodotto(const float u[], const float v[], float w[], int N) {
    // Ottengo l'indice del thread
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Trovo l'indice finale in modo che non vada fuori range
    if(i < N) w[i] = u[i] * v[i];
}


int main(int argc, char *argv[]) {
    if (argc != 2) { exit(1); }
    int N = atoi(argv[1]);
    float *du, *dv, *dw;

    // Alloco memoria
    int vec_size = N * sizeof(float);
    float *u = (float *) malloc(vec_size);
    float *v = (float *) malloc(vec_size);
    float *w = (float *) malloc(vec_size);
    hipMalloc(&du, vec_size);
    hipMalloc(&dv, vec_size);
    hipMalloc(&dw, vec_size);

    // Inizializzo i dati
    for (int i = 0; i < N; i++) {
        u[i] = (float)i;
        v[i] = (float)i;
    }

    dim3 gridDim(N/64, 1, 1);
    dim3 blockDim(64, 1, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Copio i dati
    hipMemcpy(du, u, vec_size, hipMemcpyHostToDevice);
    hipMemcpy(dv, v, vec_size, hipMemcpyHostToDevice);

    // CHIAMO KERNEL
    hipEventRecord(start);
    prodotto<<<gridDim, blockDim>>>(du, dv, dw, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Copio i dati
    hipMemcpy(w, dw, vec_size, hipMemcpyDeviceToHost);

    // Calcolo tempo richiesto
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);

    // Eseguo la somma sull'host
    float sum = 0;
    for (int i = 0; i < N; ++i) {
        sum += w[i];
    }

    // Stampe

    printf("Prodotto scalare: %.2f\n", sum);
    printf("Tempo richiesto: %f ms\n", elapsed);

    // Libero memoria
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(u);
    free(v);
    free(w);
    hipFree(du);
    hipFree(dv);
    hipFree(dw);

    return 0;
}