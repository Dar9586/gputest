#include <cstdlib>
#include <hip/hip_runtime.h>

#include <cstdio>

typedef float *floatptr;

__global__ void calc_temp(const floatptr v1, const floatptr v2, floatptr temp_res, int vec_len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    temp_res[i] += v1[i] * v2[i];
    __syncthreads();
    // Somma i risultati di tutti i thread del blocco
    for (int step = 1; step < blockDim.x; step *= 2) {
        int remainder = threadIdx.x % (step * 2);
        if (remainder == 0) {
            temp_res[i] += temp_res[i + step];
        } else {
            break;
        }
    }
}

void fill_vector(floatptr vec, int len) {
    for (int i = 0; i < len; ++i) {
        vec[i] = (float) i;
    }
}


int main(int argc, char *argv[]) {
    floatptr v1_dev, v2_dev, temp_res_dev;

    if (argc != 2) { exit(1); }
    int vec_len = atoi(argv[1]);
    // Definizione dimensione griglia
    dim3 grid_dim(vec_len / 64 + ((vec_len % 64) != 0));
    dim3 block_dim(64);
    // Creazione eventi
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocazione e inizializzazione memoria host
    floatptr v1 = (floatptr) malloc(vec_len * sizeof(float));
    floatptr v2 = (floatptr) malloc(vec_len * sizeof(float));
    floatptr temp_res = (floatptr) malloc(grid_dim.x * block_dim.x * sizeof(float));
    fill_vector(v1, vec_len);
    fill_vector(v2, vec_len);

    // Allocazione e inizializzazione memoria device
    hipMalloc(&v1_dev, vec_len * sizeof(float));
    hipMalloc(&v2_dev, vec_len * sizeof(float));
    hipMalloc(&temp_res_dev, grid_dim.x * block_dim.x * sizeof(float));
    hipMemset(temp_res_dev, 0, grid_dim.x * block_dim.x);
    hipMemcpy(v1_dev, v1, vec_len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v2_dev, v2, vec_len * sizeof(float), hipMemcpyHostToDevice);

    // Esecuzione kernel
    hipEventRecord(start);
    calc_temp<<<grid_dim, block_dim>>>(v1_dev, v2_dev, temp_res_dev, vec_len);
    // Calcolo risultato finale dai risultati parziali dei blocchi
    hipMemcpy(temp_res, temp_res_dev, grid_dim.x * block_dim.x * sizeof(float), hipMemcpyDeviceToHost);
    float total = 0;
    for (int i = 0; i < grid_dim.x; ++i) {
        total += temp_res[i * block_dim.x];
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Stampa
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("\nTotal: %f in %.4f ms\n", total, elapsed);

    // Free
    hipFree(v1_dev);
    hipFree(v2_dev);
    hipFree(v2_dev);
    free(v1);
    free(v2);
}

