#include <cstdlib>
#include <hip/hip_runtime.h>

#include <cstdio>
// 10 registri

typedef float *floatptr;

#define THREAD_PER_BLOCK 64

#define fmalloc(x) ((floatptr) malloc(fsize(x)))
#define cudafmalloc(ptr, x) (hipMalloc(&(ptr), fsize(x)))
#define fsize(x) ((x)*sizeof(float))

__global__ void calc_temp(const floatptr v1, const floatptr v2, floatptr res, int vec_len) {
    extern __shared__ float temp_res_shared[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= vec_len)return;
    temp_res_shared[threadIdx.x] = v1[i] * v2[i];
    __syncthreads();
    // Somma i risultati di tutti i thread del blocco
    for (int step = blockDim.x / 2; step > 0; step /= 2) {
        if (threadIdx.x < step)
            temp_res_shared[threadIdx.x] += temp_res_shared[threadIdx.x + step];
        __syncthreads();
    }
    if (threadIdx.x == 0)
        res[blockIdx.x] = temp_res_shared[threadIdx.x];
}

void fill_vector(floatptr vec, int len) {
    for (int i = 0; i < len; ++i) {
        vec[i] = (float) i;
    }
}

int main(int argc, char *argv[]) {
    floatptr v1_dev, v2_dev, temp_res_dev;
    if (argc != 2) { exit(1); }
    int vec_len = atoi(argv[1]);
    // Definizione dimensione griglia
    dim3 grid_dim(vec_len / THREAD_PER_BLOCK + ((vec_len % THREAD_PER_BLOCK) != 0));
    dim3 block_dim(THREAD_PER_BLOCK);
    unsigned long shared_mem = fsize(block_dim.x);

    // Creazione eventi
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocazione e inizializzazione memoria host
    floatptr v1 = fmalloc(vec_len);
    floatptr v2 = fmalloc(vec_len);
    floatptr temp_res = fmalloc(grid_dim.x);

    fill_vector(v1, vec_len);
    fill_vector(v2, vec_len);

    // Allocazione e inizializzazione memoria device
    cudafmalloc(v1_dev, vec_len);
    cudafmalloc(v2_dev, vec_len);
    cudafmalloc(temp_res_dev, grid_dim.x);

    hipMemcpy(v1_dev, v1, fsize(vec_len), hipMemcpyHostToDevice);
    hipMemcpy(v2_dev, v2, fsize(vec_len), hipMemcpyHostToDevice);

    // Esecuzione kernel
    hipEventRecord(start);
    calc_temp<<<grid_dim, block_dim, shared_mem>>>(v1_dev, v2_dev, temp_res_dev, vec_len);
    // Calcolo risultato finale dai risultati parziali dei blocchi
    hipMemcpy(temp_res, temp_res_dev, fsize(grid_dim.x), hipMemcpyDeviceToHost);
    float total = 0;
    for (int i = 0; i < grid_dim.x; ++i) {
        total += temp_res[i];
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Stampa
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("\nTotal: %f in %.4f ms\n", total, elapsed);

    // Free
    hipFree(v1_dev);
    hipFree(v2_dev);
    hipFree(temp_res_dev);
    free(v1);
    free(v2);
    free(temp_res);
}
