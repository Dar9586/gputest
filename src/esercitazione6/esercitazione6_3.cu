#include <cstdlib>
#include <hip/hip_runtime.h>

#include <cstdio>

typedef float *floatptr;

__global__ void calc_temp(const floatptr v1, const floatptr v2, floatptr res, int vec_len) {
    extern __shared__ int temp_res_shared[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    temp_res_shared[threadIdx.x] = v1[i] * v2[i];
    __syncthreads();
    // Somma i risultati di tutti i thread del blocco
    for (int step = blockDim.x / 2; step > 0; step /= 2) {
        if (threadIdx.x < step)
            temp_res_shared[threadIdx.x] += temp_res_shared[threadIdx.x + step];
    }
    if(threadIdx.x==0)
        res[blockIdx.x]=temp_res_shared[threadIdx.x];
}

void fill_vector(floatptr vec, int len) {
    for (int i = 0; i < len; ++i) {
        vec[i] = (float) i;
    }
}


int main(int argc, char *argv[]) {
    floatptr v1_dev, v2_dev, temp_res_dev;

    if (argc != 2) { exit(1); }
    int vec_len = atoi(argv[1]);
    // Definizione dimensione griglia
    dim3 grid_dim(vec_len/64+((vec_len%64)!=0));
    dim3 block_dim(64);
    printf("Grid %d, blocks %d\n",grid_dim.x,block_dim.x);
    // Creazione eventi
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocazione e inizializzazione memoria host
    floatptr v1 = (floatptr) malloc(vec_len * sizeof(float));
    floatptr v2 = (floatptr) malloc(vec_len * sizeof(float));
    floatptr temp_res = (floatptr) malloc(grid_dim.x * sizeof(float));

    fill_vector(v1, vec_len);
    fill_vector(v2, vec_len);

    // Allocazione e inizializzazione memoria device
    hipMalloc(&v1_dev, vec_len * sizeof(float));
    hipMalloc(&v2_dev, vec_len * sizeof(float));
    hipMalloc(&temp_res_dev, grid_dim.x * sizeof(float));

    hipMemcpy(v1_dev, v1, vec_len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v2_dev, v2, vec_len * sizeof(float), hipMemcpyHostToDevice);

    // Esecuzione kernel
    hipEventRecord(start);
    calc_temp<<<grid_dim, block_dim,block_dim.x>>>(v1_dev, v2_dev, temp_res_dev, vec_len);
    // Calcolo risultato finale dai risultati parziali dei blocchi
    hipMemcpy(temp_res, temp_res_dev, grid_dim.x * sizeof(float), hipMemcpyDeviceToHost);
    float total = 0;
    for (int i = 0; i < grid_dim.x; ++i) {
        printf("%.1f, ",temp_res[i]);
        total += temp_res[i];
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Stampa
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("\nTotal: %f in %.4f ms\n", total,elapsed);

    // Free
    hipFree(v1_dev);
    hipFree(v2_dev);
    hipFree(v2_dev);
    free(v1);
    free(v2);
}

