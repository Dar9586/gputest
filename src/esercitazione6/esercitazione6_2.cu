#include <cstdlib>
#include <hip/hip_runtime.h>

#include <cstdio>

typedef float *floatptr;

__global__ void calc_temp(const floatptr v1, const floatptr v2, floatptr res,floatptr out, int vec_len) {
    extern __shared__ int temp_res_shared[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    out[i]=temp_res_shared[threadIdx.x] = v1[i] * v2[i];

    __syncthreads();
    // Somma i risultati di tutti i thread del blocco
    for (int step = 1; step < blockDim.x; step *= 2) {
        int remainder = threadIdx.x % (step * 2);
        if (remainder == 0) {
            temp_res_shared[threadIdx.x] += temp_res_shared[threadIdx.x + step];
        } else {
            break;
        }
    }
    if(threadIdx.x==0)
        res[blockIdx.x]=temp_res_shared[0];
}

void fill_vector(floatptr vec, int len) {
    for (int i = 0; i < len; ++i) {
        vec[i] = (float) i;
    }
}


int main(int argc, char *argv[]) {
    floatptr v1_dev, v2_dev, temp_res_dev,out_dev;

    if (argc != 2) { exit(1); }
    int vec_len = atoi(argv[1]);
    // Definizione dimensione griglia
    dim3 grid_dim(vec_len/64+((vec_len%64)!=0));
    dim3 block_dim(64);
    printf("Grid %d, blocks %d\n",grid_dim.x,block_dim.x);
    // Creazione eventi
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocazione e inizializzazione memoria host
    floatptr v1 = (floatptr) malloc(vec_len * sizeof(float));
    floatptr v2 = (floatptr) malloc(vec_len * sizeof(float));
    floatptr out = (floatptr) malloc(vec_len * sizeof(float));
    floatptr temp_res = (floatptr) malloc(grid_dim.x * sizeof(float));

    fill_vector(v1, vec_len);
    fill_vector(v2, vec_len);

    // Allocazione e inizializzazione memoria device
    hipMalloc(&v1_dev, vec_len * sizeof(float));
    hipMalloc(&v2_dev, vec_len * sizeof(float));
    hipMalloc(&out_dev, vec_len * sizeof(float));
    hipMalloc(&temp_res_dev, grid_dim.x * sizeof(float));

    hipMemcpy(v1_dev, v1, vec_len * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(v2_dev, v2, vec_len * sizeof(float), hipMemcpyHostToDevice);

    // Esecuzione kernel
    hipEventRecord(start);
    calc_temp<<<grid_dim, block_dim,block_dim.x*sizeof(float)>>>(v1_dev, v2_dev, temp_res_dev,out_dev, vec_len);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

    // Calcolo risultato finale dai risultati parziali dei blocchi
    hipMemcpy(temp_res, temp_res_dev, grid_dim.x * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(out, out_dev, vec_len * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < 64; ++i) {
        printf("%d:%.1f, ",i,out[(91*block_dim.x)+ i]);
    }
    printf("\n");
    float total = 0;
    for (int i = 0; i < grid_dim.x; ++i) {
        printf("%d:%.1f, \n",i,temp_res[i]);
        total += temp_res[i];
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Stampa
    float elapsed;
    hipEventElapsedTime(&elapsed, start, stop);
    printf("\nTotal: %f in %.4f ms\n", total,elapsed);

    // Free
    hipFree(v1_dev);
    hipFree(v2_dev);
    hipFree(v2_dev);
    free(v1);
    free(v2);
}

